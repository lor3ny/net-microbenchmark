#include "hip/hip_runtime.h"
#include <mpi.h>
#include <iostream>
#include <cstring>
#include <climits>
#include <cassert>
#include <hip/hip_runtime.h>

using namespace std;

#define MiB1 1048576
#define WARM_UP 10
#define BENCHMARK_ITERATIONS 100

// TO TEST THE FINAL VERSION
//https://github.com/NVIDIA/nccl-tests

static int smallest_negabinary[20] = {0, 0, -2, -2, -10, -10, -42, -42,
  -170, -170, -682, -682, -2730, -2730, -10922, -10922, -43690, -43690, -174762, -174762};
static int largest_negabinary[20] = {0, 1, 1, 5, 5, 21, 21, 85, 85,
  341, 341, 1365, 1365, 5461, 5461, 21845, 21845, 87381, 87381, 349525};


#define CUDA_CHECK(cmd) do {                        \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


inline int pi(int rank, int step, int comm_sz) {
  int dest;

  if((rank & 1) == 0) dest = (int)((rank + (1-1*pow(-2,step+1))/3) + comm_sz) % comm_sz;//(rank + rhos[step]) % comm_sz;  // Even rank
  else dest = (int)((rank - (1-1*pow(-2,step+1))/3) + comm_sz) % comm_sz; //(rank - rhos[step]) % comm_sz;                 // Odd rank

  if(dest < 0) dest += comm_sz;                              // Adjust for negative ranks

  return dest;
}

inline int log_2(int value) {
  if (1 > value) {
      return -1;
  }
  return sizeof(int)*8 - 1 - __builtin_clz(value);
}

static inline uint32_t reverse(uint32_t x){
  x = ((x >> 1) & 0x55555555u) | ((x & 0x55555555u) << 1);
  x = ((x >> 2) & 0x33333333u) | ((x & 0x33333333u) << 2);
  x = ((x >> 4) & 0x0f0f0f0fu) | ((x & 0x0f0f0f0fu) << 4);
  x = ((x >> 8) & 0x00ff00ffu) | ((x & 0x00ff00ffu) << 8);
  x = ((x >> 16) & 0xffffu) | ((x & 0xffffu) << 16);
  return x;
}

static uint32_t binary_to_negabinary(int32_t bin) {
  assert(bin <= 0x55555555);
  const uint32_t mask = 0xAAAAAAAA;
  return (mask + bin) ^ mask;
}

static inline int in_range(int x, uint32_t nbits){
  return x >= smallest_negabinary[nbits] && x <= largest_negabinary[nbits];
}

static inline uint32_t get_rank_negabinary_representation(uint32_t num_ranks, uint32_t rank){
  binary_to_negabinary(rank);
  uint32_t nba = UINT32_MAX, nbb = UINT32_MAX;
  size_t num_bits = log_2(num_ranks);
  if(rank % 2){
      if(in_range(rank, num_bits)){
          nba = binary_to_negabinary(rank);
      }
      if(in_range(rank - num_ranks, num_bits)){
          nbb = binary_to_negabinary(rank - num_ranks);
      }
  }else{
      if(in_range(-rank, num_bits)){
          nba = binary_to_negabinary(-rank);
      }
      if(in_range(-rank + num_ranks, num_bits)){
          nbb = binary_to_negabinary(-rank + num_ranks);
      }
  }

  assert(nba != UINT32_MAX || nbb != UINT32_MAX);

  if(nba == UINT32_MAX && nbb != UINT32_MAX){
      return nbb;
  }else if(nba != UINT32_MAX && nbb == UINT32_MAX){
      return nba;
  }else{ // Check MSB
      if(nba & (80000000 >> (32 - num_bits))){
          return nba;
      }else{
          return nbb;
      }
  }
}


inline uint32_t remap_rank(uint32_t num_ranks, uint32_t rank){
  uint32_t remap_rank = get_rank_negabinary_representation(num_ranks, rank);    
  remap_rank = remap_rank ^ (remap_rank >> 1);
  size_t num_bits = log_2(num_ranks);
  remap_rank = reverse(remap_rank) >> (32 - num_bits);
  return remap_rank;
}


__global__ void reduce_sum_kernel(const int *in, int *inout, size_t count) {

  int global_thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int thread_count = gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
  int i, idx;
  for(i = 0; global_thread_idx + i*thread_count < count; i++){
    idx = global_thread_idx + i*thread_count; 
    inout[idx] += in[idx]; 
  }

  /*
  for(int i = 0; i<count; i++){
    inout[i] += in[i];    
  }
  */
}

int allreduce_swing_bdw_remap(const void *send_buf, void *recv_buf, size_t count,
  MPI_Datatype dtype, MPI_Op op, MPI_Comm comm){

  int size, rank, dest, steps, step, datatype_size, err = MPI_SUCCESS;
  int *r_count = NULL, *s_count = NULL, *r_index = NULL, *s_index = NULL;
  size_t w_size;
  uint32_t vrank, vdest;

  char *tmp_send = NULL, *tmp_recv = NULL;
  char *tmp_buf_raw = NULL, *tmp_buf;
  ptrdiff_t lb, extent, true_extent, gap = 0, buf_size;

  MPI_Comm_size(comm, &size);
  MPI_Comm_rank(comm, &rank);
  MPI_Type_size(dtype, &datatype_size);

  // Does not support non-power-of-two or negative sizes
  steps = log_2(size);

  // Allocate temporary buffer for send/recv and reduce operations
  MPI_Type_get_extent(dtype, &lb, &extent);
  MPI_Type_get_true_extent(dtype, &gap, &true_extent);
  buf_size = true_extent + extent * (count >> 1);
  //tmp_buf_raw = (char *)malloc(buf_size);
  CUDA_CHECK(hipMalloc((void**) &tmp_buf_raw, buf_size));
  //CUDA_CHECK(hipMemset(tmp_+buf, 0, buf_size));
  tmp_buf = tmp_buf_raw - gap;

  // Copy into receive_buffer content of send_buffer to not produce
  // side effects on send_buffer
  if(send_buf != MPI_IN_PLACE) {
    CUDA_CHECK(hipMemcpy(recv_buf, send_buf, count * datatype_size, hipMemcpyDeviceToDevice));
  }


  //CUDA_CHECK(hipMalloc((void**) &r_index, sizeof(*r_index) * steps);
  //CUDA_CHECK(hipMalloc((void**) &s_index, sizeof(*s_index) * steps);
  //CUDA_CHECK(hipMalloc((void**) &r_count, sizeof(*r_count) * steps);
  //CUDA_CHECK(hipMalloc((void**) &s_count, sizeof(*s_count) * steps);
  r_index = (int*) malloc(sizeof(*r_index) * steps);
  s_index = (int*) malloc(sizeof(*s_index) * steps);
  r_count = (int*) malloc(sizeof(*r_count) * steps);
  s_count = (int*) malloc(sizeof(*s_count) * steps);
  if(NULL == r_index || NULL == s_index || NULL == r_count || NULL == s_count) {
    err = MPI_ERR_NO_MEM;
    goto cleanup_and_return;
  }

  w_size = count;
  s_index[0] = r_index[0] = 0;
  vrank = remap_rank((uint32_t) size, (uint32_t) rank);

  // Reduce-Scatter phase
  CUDA_CHECK(hipDeviceSynchronize());

  for(step = 0; step < steps; step++) {
    dest = pi(rank, step, size);
    vdest = remap_rank((uint32_t) size, (uint32_t) dest);

    if(vrank < vdest) {
      r_count[step] = w_size / 2;
      s_count[step] = w_size - r_count[step];
      s_index[step] = r_index[step] + r_count[step];
    } else {
      s_count[step] = w_size / 2;
      r_count[step] = w_size - s_count[step];
      r_index[step] = s_index[step] + s_count[step];
    }
    tmp_send = (char *)recv_buf + s_index[step] * extent;
    err = MPI_Sendrecv(tmp_send, s_count[step], dtype, dest, 0,
                        tmp_buf, r_count[step], dtype, dest, 0,
                        comm, MPI_STATUS_IGNORE);
    if(MPI_SUCCESS != err) { goto cleanup_and_return; }

    tmp_recv = (char *) recv_buf + r_index[step] * extent;
    
    //MPI_Reduce_local(tmp_buf, tmp_recv, r_count[step], dtype, op);
    reduce_sum_kernel<<<512, 512>>>((const int*)tmp_buf, (int*)tmp_recv, r_count[step]);
    //reduce_sum_kernel<<<1, 1>>>((const int*)tmp_buf, (int*)tmp_recv, r_count[step]);

    if(step + 1 < steps) {
      r_index[step + 1] = r_index[step];
      s_index[step + 1] = r_index[step];
      w_size = r_count[step];
    }

    CUDA_CHECK(hipDeviceSynchronize());

  }

  // Allgather phase
  for(step = steps - 1; step >= 0; step--) {
    dest = pi(rank, step, size);

    tmp_send = (char *)recv_buf + r_index[step] * extent;
    tmp_recv = (char *)recv_buf + s_index[step] * extent;
    err = MPI_Sendrecv(tmp_send, r_count[step], dtype, dest, 0,
                      tmp_recv, s_count[step], dtype, dest, 0,
                      comm, MPI_STATUS_IGNORE);
    if(MPI_SUCCESS != err) { goto cleanup_and_return; }

    CUDA_CHECK(hipDeviceSynchronize());
  }

  CUDA_CHECK(hipFree(tmp_buf_raw));
  free(r_index);
  free(s_index);
  free(r_count);
  free(s_count);
  return MPI_SUCCESS;

  cleanup_and_return:
  if(NULL != tmp_buf_raw)  CUDA_CHECK(hipFree(tmp_buf_raw));
  if(NULL != r_index)      free(r_index);
  if(NULL != s_index)      free(s_index);
  if(NULL != r_count)      free(r_count);
  if(NULL != s_count)      free(s_count);
  return err;
}


int VerifyCollective(int* buf_a, int* buf_b, int dim, int rank){
  int incorrect = 0;
  for(int i = 0; i<dim; ++i){
    try {
      if(buf_a[i] != buf_b[i]){
        cout << rank << " : "<< i <<" - cuda: "<< buf_a[i] << " test: " << buf_b[i] << endl;
        incorrect = -1;
      }
    } catch (const invalid_argument& e) {
        cerr << "ERROR: Memory corruption on verification." << endl;
        return EXIT_FAILURE;
    }
  }
  return incorrect;
}



int main(int argc, char *argv[]) {
    MPI_Init(&argc, &argv);

    int rank, size, name_len, ret;
    double total_time = 0.0;
    char processor_name[MPI_MAX_PROCESSOR_NAME];
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    MPI_Get_processor_name(processor_name, &name_len);

    if (argc < 2) {
        cerr << "Please, insert an integer as argument" << endl;
        return 1;  
    }

    int mib_count = 0;
    try {
      mib_count = stoi(argv[1]);  
      if(rank == 0)
        cout << endl << "Message is " << mib_count << " MiB - ALL REDUCE" << endl;
    } catch (const invalid_argument& e) {
        cout << "Not valid argument!" << endl;
        return EXIT_FAILURE;
    }
    MPI_Barrier(MPI_COMM_WORLD);
    cout << " {" << rank << " : "<< processor_name << "}" << endl;

    int msg_count = (mib_count * MiB1)/sizeof(int);
    int BUFFER_SIZE = (mib_count * MiB1);
    int *h_send_buffer = (int*) malloc(BUFFER_SIZE); 
    int *h_recv_buffer = (int*) malloc(BUFFER_SIZE);
    int *h_test_recv_buffer = (int*) malloc(BUFFER_SIZE);

    // BISOGNA SETTARE LA GPU, FORSE BISOGNA FARLO SEMPRE
    int gpu_rank = rank % 1;
    CUDA_CHECK(hipSetDevice(gpu_rank));

    int *d_send_buffer;
    CUDA_CHECK(hipMalloc((void**)&d_send_buffer, (size_t) BUFFER_SIZE));
    int *d_recv_buffer;
    CUDA_CHECK(hipMalloc((void**)&d_recv_buffer, (size_t) BUFFER_SIZE));
    int *d_test_recv_buffer;
    CUDA_CHECK(hipMalloc((void**)&d_test_recv_buffer, (size_t) BUFFER_SIZE));

    

    for (int i = 0; i < msg_count; i++) {
        h_send_buffer[i] = (float) rank; 
    }
    CUDA_CHECK(hipMemcpy(d_send_buffer, h_send_buffer, (size_t) BUFFER_SIZE, hipMemcpyHostToDevice));

    allreduce_swing_bdw_remap(d_send_buffer, d_recv_buffer, (size_t) msg_count, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
    MPI_Allreduce(d_send_buffer, d_test_recv_buffer, (size_t) msg_count, MPI_INT, MPI_SUM, MPI_COMM_WORLD);

    CUDA_CHECK(hipMemcpy(h_recv_buffer, d_recv_buffer, (size_t) BUFFER_SIZE, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_test_recv_buffer, d_test_recv_buffer, (size_t) BUFFER_SIZE, hipMemcpyDeviceToHost));
  
    ret = VerifyCollective(h_recv_buffer, h_test_recv_buffer, BUFFER_SIZE/sizeof(int), rank);
    if(ret==-1){
      cerr << "THE ANALYZED COLLECTIVE IS NOT WORKING! :(" << endl;
      free(h_send_buffer);
      free(h_recv_buffer);
      free(h_test_recv_buffer);

      CUDA_CHECK(hipFree(d_recv_buffer));
      CUDA_CHECK(hipFree(d_send_buffer));
      CUDA_CHECK(hipFree(d_test_recv_buffer));
      return EXIT_FAILURE;
    }


    MPI_Barrier(MPI_COMM_WORLD);
    for(int i = 0; i < BENCHMARK_ITERATIONS + WARM_UP; ++i){

        double start_time = MPI_Wtime();
        allreduce_swing_bdw_remap(d_send_buffer, d_recv_buffer, (size_t) msg_count, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
        double end_time = MPI_Wtime();

        if(i>WARM_UP) {
            total_time += end_time - start_time;
        }

        MPI_Barrier(MPI_COMM_WORLD);
    }
    total_time = (double)(total_time)/BENCHMARK_ITERATIONS;

    double max_time;
    MPI_Reduce(&total_time, &max_time, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);

    CUDA_CHECK(hipMemcpy(h_recv_buffer, d_recv_buffer, (size_t) BUFFER_SIZE, hipMemcpyDeviceToHost));

    uint64_t verifier = 0;
    for(int i = 0; i<msg_count; i++){
      verifier += h_recv_buffer[i];
    }

    MPI_Barrier(MPI_COMM_WORLD);

    if(rank == 0){
      float buffer_gib = (BUFFER_SIZE / (float) (1024*1024*1024)) * 8;
      float bandwidth =  2 * buffer_gib * ((size-1)/(float)size);
      bandwidth = bandwidth / max_time;
      cout << "Buffer: "  << BUFFER_SIZE << " byte - " << buffer_gib << " Gib - " << mib_count << " MiB, verifier: " << verifier << ", Latency: " << max_time << ", Bandwidth: " << bandwidth << endl;
    }

    free(h_send_buffer);
    free(h_recv_buffer);
    free(h_test_recv_buffer);

    CUDA_CHECK(hipFree(d_recv_buffer));
    CUDA_CHECK(hipFree(d_send_buffer));
    CUDA_CHECK(hipFree(d_test_recv_buffer));

    MPI_Finalize();
    return EXIT_SUCCESS;
}

